#include "hip/hip_runtime.h"
/*
* psf_neural_discover.cu is a program to take matlab images and process them through a neural network to determine activation values for pixels in a 7x7 region centered on each pixel
*  We are assuming a single output for the neural network, and a single hidden layer of 100 nodes this will perform the necessary calculation on each pixel in the image and return
*	an image of 0 or 1 to be used later in the computation
*	V 1.0
*		we expect a format of [im_activate] = image_neural [i1, theta1, threta2, numoframes];
*	AJN 11/2/15
*/

#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#define PI 3.14159265358979323846
#define O_TILE_WIDTH 20								// variable to determine how many output tiles will be considered in a block
# define BLOCK_WIDTH (O_TILE_WIDTH + (9-1))		// block width needs to be output tiles + mask_width - 1 to ensure enough pixels are covered for calculation




/*
* Device code
*
*
*/



void __global__ activate(float *d_iall,   // the gaussian is a separable filter and be treated as such
	float *d_theta1,	// makes these elements eligible for constant caching
	float *d_theta2,
	float *d_ifin,
	int irow,
	int icol,
	int numi)
{
	// Declare variables
	//float d_i3[(50)] = { 0 };
	__shared__ float d_i2[(BLOCK_WIDTH)][(BLOCK_WIDTH)];		// preallocate space for shared image
	/*__shared__ float d_th1[(50)][(100)];
	__shared__ float d_th2[(100)];
*/
	// Coordinate building
	int tx = threadIdx.x;			// local x coord
	int ty = threadIdx.y;			// local y coord
	int tz = threadIdx.z;
	// location of output pixel being analyzed
	int row_output = blockIdx.x*O_TILE_WIDTH + tx;		// gives y coordinate as a function of tile width    **these lose meaning for (ty || tx) >= O_TILE_WIDTH and the same is true for **
	int col_output = blockIdx.y*O_TILE_WIDTH + ty;		// gives x coordinate as a function of tile width
	int imnum = blockIdx.z;
	if (imnum < numi){
	// initialize location of apron		this forces the first pixel to take care of both the first output pixel, and loading the first input pixel
	// BLOCK_WIDTH is larger than O_TILE_WIDTH so there are more threads being used than output pixels being calculated
	int row_input = row_output - 4;	// EACH thread should load 1 input tile to the shared image as there are [BLOCK_WIDTH]x[BLOCK_WIDTH] threads in a block
	int col_input = col_output - 4;	// and BLOCK_WIDTH = O_TILE_WIDTH + MASK_WIDTH-1
/*
	// Buffer data into block
	for (int tcol = 0; tcol < 100; tcol++){   // buffer theta1 matrix into shared block space
		for (int trow = 0; trow < 50; trow++){
			d_th1[trow][tcol] = d_theta1[trow + 50 * tcol];
		}
	}

	// Buffer theta2 into shared block space
	for (int t2row = 0; t2row < 101; t2row++){
		d_th2[t2row] = d_theta2[t2row];
	} */
	// buffer shared image into d_i2															
	// row/col_input represents the row/col of the input pixel being considered by 
	// thread [blockIdx.y*BLOCK_WIDTH+ty][blockIdx.x*BLOCK_WIDTH+tx]
	if ((row_input >= 0) && (row_input < irow) && (col_input >= 0) && (col_input < icol)){		// if statement checks the row/col indices to ensure they fall onto the input image
		d_i2[ty][tx] = d_iall[row_input + col_input*irow + imnum*irow*icol];										// if true, the value of the image is written to the shared array at location d_i2[ty][tx] and stored locally
	}																							// on the block
	else{
		d_i2[ty][tx] = 0;																	// If row/col do not satisfy boundary condtions then assign a 0 to the value to build and apron of 
	}																							// of pixels that will not contribute to the calculation

	__syncthreads();																			// each thread uploads to a shared array later accessed by all threads, it is imperative to synch threads here
	//d_i3[0] = 1.0;
	// convolution calculation
	float z1[(30)] = { 0 };
	float a1[(31)] = { 0 };
	float a = 0.0;
	float z2 = 0.0;
	a1[0] = 1;
	if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {										// check that the local thread should be apart of the calcualtion
		/*for (int rowcount = 0; rowcount < 7; rowcount++){
			for (int colcount = 0; colcount < 7; colcount++){
				d_i3[rowcount + 7 * colcount + 1] = d_i2[rowcount + ty][colcount + tx];					// linearize region to prep for matrix math ensure coloumn major setup for proper neural net behavior
			} // end coloumn image for loop
		}*/
		// At this point d_i3 should be linearized as matlab would do so
		// perform matrix calculation here
		for (int th1count = 0; th1count < 30; th1count++){
			z1[th1count] = d_theta1[82*th1count];
			for (int rowcount = 0; rowcount < 9; rowcount++){
				for (int colcount = 0; colcount < 9; colcount++){
					z1[th1count] += d_i2[rowcount+ty][colcount + tx] * d_theta1[rowcount + 9*colcount + 1 + 82*th1count];
				} 
			}
			a1[th1count + 1] = powf(1.0 + exp(-z1[th1count]), -1.0);
		} // this completes the first half of the calculation

		for (int th2count = 0; th2count<31; th2count++){
			z2 += a1[th2count] * d_theta2[th2count];
		}
		a = powf(1 + exp(-z2), -1.0); // activation value
		//a = z2;


		if (row_output < irow && col_output < icol && imnum < numi){

			d_ifin[row_output + col_output*irow + imnum*irow*icol] = tx;			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin

		}
		/*else{
		__syncthreads();
		d_ifin[row_output + col_output*irow + imnum*irow*icol] = d_ifin[row_output + col_output*irow];			// assign to output variable  THIS SECTION WILL CORRECTLY WRITE TO d_ifin
		__syncthreads();
		}  // end if else statement tyo decide what to write to final image
		*/
	}  // end if statement to decide whether to calculate output
	}
} // end gpu  void activate 




/*

THIS IS THE SECTION FOR IDENTIFICATION

*/





/*
* Host code
*
*
*/


void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, mxArray const *prhs[])
{
	/* Declare all variables.*/
	float *iall;			// the pointer to the array of all images to be analyzed
	float *theta1;			// pointer to theta1 matrix
	float *theta2;			// pointer to theta2 matrix
	float  *d_iall;		// Pointer to image array on gpu
	float *d_theta1;		// Pointer to d_theta1 on gpu
	float *d_theta2;		// Pointer to d_theta2 on gpu
	float *d_ifin;			// pointer to d_ifin on gpu
	int irow;				// number of pixels in a row which should also be the number in a coloumn
	int icol;				// n
	int numi;				// number of images imported
	const size_t *idims, *th1dims, *th2dims;


	/* Throw an error if the input does not match expectations. */
/*	if (nrhs != 4) {
		printf("Must have 4 inputs ( i1, theta1, theta2, numthreads) line: %d\n", __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	if (!mxIsfloat(prhs[0]) || mxIsComplex(prhs[0])){
		printf("i1 must be a n x m x numel(i1(1,1,:)) float array\n");
		mexErrMsgTxt("See Error above!\n");

	}
	if (!mxIsfloat(prhs[1]) || mxIsComplex(prhs[1])){
		printf("Theta1 must be a n +1 x l float array\n");
		mexErrMsgTxt("See Error above!\n");
	}
	if (!mxIsfloat(prhs[2]) || mxIsComplex(prhs[2])){
		printf("Theta2 must be a l x 1 float array\n");
		mexErrMsgTxt("See Error above!\n");
	}
	if (!mxIsfloat(prhs[3]) || mxIsComplex(prhs[3])){
		printf("number of threads per block must be an integer between 1 and 1024\n");
		mexErrMsgTxt("See Error above!\n");
	}
*/

	// get pointer to input arguments
	iall = (float *)mxGetPr(prhs[0]);		// matlab linearizes in a coloumn major format which affects indexing (Writing MAtlab C/MEX Code - Research Gate)
	idims = mxGetDimensions(prhs[0]);	// get dimensions of image array
	icol = (int)idims[1];
	irow = (int)idims[0];
	numi = mxGetScalar(prhs[3]);  // get number of images perblock from matlab


	// get theta1 dims
	theta1 = (float *)mxGetPr(prhs[1]);
	th1dims = mxGetDimensions(prhs[1]);
	int th1row = (int)th1dims[0]; // number of rows in theta1
	int th1col = (int)th1dims[1]; // number of coloumns in theta1

	// get theta2 dims
	theta2 = (float *)mxGetPr(prhs[2]);
	th2dims = mxGetDimensions(prhs[2]);
	int th2row = (int)th2dims[0]; // number of rows in theta2
	int th2col = 1; // number of coloumns in theta2

/*
	// EVERYONE LOVES SOME GOOD VARIABLE CHECKING!!!!!!!!!!!!!!
	if (th1row != 50){
		printf("Theta1 must have 50 rows for what you want to do\n");
		mexErrMsgTxt("See Above Error!\n");
	}

	if (th1col != 30 || th2row != 31){
		printf("Theta2 must have oone more row than Theta1 has coloumns and that number should be 100\n");
		mexErrMsgTxt("See Above Error!\n");
	}

	if (th2col != 1){
		printf("Theta2 must have 1 coloumn\n");
		mexErrMsgTxt("See Above Error!\n");
	}

	// Did the User declare an output?
	if (nlhs != 1){
		printf("Declare an output variable [im_activate] = image_neural(i1, Theta1.', Theta2.')\n"); // oh user...... TEACH THEM A LESSON!!!!
		mexErrMsgTxt("See Error above!\n");
	}*/
	hipDeviceReset();
	// allocate memory on the gpu device
	hipError_t err1 = hipMalloc((void**)&d_iall, irow*icol*(numi)*sizeof(float));				// allocate image memory
	if (err1 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err2 = hipMalloc((void**)&d_theta1, th1row*th1col*sizeof(float));						// allocate theta1 memory
	if (err2 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err3 = hipMalloc((void**)&d_theta2, th2row*sizeof(float));						// allocate theta2 memory
	if (err3 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err4 = hipMalloc((void**)&d_ifin, irow*icol*(numi)*sizeof(float));						// allocate completed activation image memory this will be a float for convience
	if (err4 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err4), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	// copy data from host to device
	hipError_t err6 = hipMemcpy(d_iall, iall, irow*icol*(numi)*sizeof(float), hipMemcpyHostToDevice);	// copy image data to gpu
	if (err6 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err6), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err7 = hipMemcpy(d_theta1, theta1, th1row*th1col*sizeof(float), hipMemcpyHostToDevice);		// copy theta1 data to gpu
	if (err7 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err7), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}

	hipError_t err8 = hipMemcpy(d_theta2, theta2, th2row*sizeof(float), hipMemcpyHostToDevice);		// copy theta2 data to gpu
	if (err8 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err8), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
/*
	hipError_t err10 = hipMemcpy(d_ifin, iall, irow*icol*numi*sizeof(float), hipMemcpyHostToDevice);	// copy image data to gpu
	if (err10 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err10), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
*/
	/* Run GPU kernel*/
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH); // run 2-D gpu kernel to help with indexing
	dim3 dimGrid((irow - 1) / O_TILE_WIDTH + 1, (icol - 1) / O_TILE_WIDTH + 1, numi );
	



	//printf("numi = %d, irow = %d, icol = %d, th1row = %d, th1col = %d, th2row = %d, th2col = %d\n", numi, irow, icol, th1row, th1col, th2row, th2col);
	activate << < dimGrid, dimBlock >> >(d_iall, d_theta1, d_theta2, d_ifin, irow, icol, (numi));

	//ident << < dimGrid, dimBlock >> >(d_ifin, d_iout, irow, icol, numi);

	/*		 copy data back to mxarray pointers for output
	*
	*
	*		Duplicate the input array of equal size to the output array
	*		Send the pointer to a variable
	*		copy data to place pointer points to, which is output
	*/

/*
	hipError_t errk1 = hipPeekAtLastError();
	if (errk1 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(errk1), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	*/
	hipError_t errk2 = hipDeviceSynchronize();
	if (errk2 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(errk2), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}


	plhs[0] = mxDuplicateArray(prhs[0]);
	float *iout = (float *)mxGetPr(plhs[0]);
	//printf("%d\n",numi);
	hipError_t err9 = hipMemcpy(iout, d_ifin, irow*icol*(numi)*sizeof(float), hipMemcpyDeviceToHost);	// copy xf_all data
	if (err9 != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err9), __FILE__, __LINE__);
		mexErrMsgTxt("See Error above!\n");
	}
	//	printf("irow %f, icol %f, numi %f, line %d\n", ifin[0], ifin[1], ifin[2], __LINE__);

	hipDeviceReset();
/*
	hipFree(d_iall);
	hipFree(d_theta1);
	hipFree(d_theta2);
	hipFree(d_ifin);
*/
	return;
}